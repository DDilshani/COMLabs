
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define SIZE 8

__global__ void addArray(double * result, double * array);

int main(){

	hipEvent_t start,stop;
	float elapsedtime;
	
	//the moment at which we start measuring the time
	hipEventCreate(&start);
	hipEventRecord(start,0);	

	double array[SIZE];
	double result[SIZE/2];
	int i;

	for (i = 0; i < SIZE; i++){
		array[i] = i +1 ;
		//printf("%lf ", array[i] );
	}

	//pointers to the arrays to be put in cuda memory
	double *array_cuda;
	double *result_cuda;

	//allocate memory in cuda device
	hipMalloc((void **)&array_cuda, sizeof(double)* SIZE);
	hipMalloc((void **)&result_cuda, sizeof(double)* (SIZE/2));

	//Copy contents from main memory to device memory
	hipMemcpy(array_cuda, array, sizeof(double)*SIZE, hipMemcpyHostToDevice);

	//call the cuda kernel
	addArray <<< 1, SIZE/2 >>> (result_cuda, array_cuda);

	//Copy results from device to host
	hipMemcpy(result, result_cuda, sizeof(double)* (SIZE/2), hipMemcpyDeviceToHost);

	printf("Answer is : ");

	for(i=0; i<SIZE/2 ;i++){
		printf("%.1lf ",result[i]);
	}

	//the moment at which we stop measuring time 
	hipEventCreate(&stop);
	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	
	//Find and print the elapsed time
	hipEventElapsedTime(&elapsedtime,start,stop);
	printf("Time spent for operation is %.10f seconds\n",elapsedtime/(float)1000);
	
	return 0;

}

__global__ void addArray(double *result_cuda, double *array_cuda){
	
	int tid = threadIdx.x;
	result_cuda[tid] = (array_cuda[2*tid] + array_cuda[2*tid+1]) / 2;
}
